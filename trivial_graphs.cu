// Parallel Adjacency Matrix Initialization

#include <hip/hip_runtime.h>
#include <iostream>
using namespace std;

__global__ void initializeGraph( int* graph, int V ){
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if( idx < V * V ){
        graph[idx] = 0;
    }
}

int main(){
    int V = 3;
    int* d_graph;
    size_t size = V * V * sizeof( int );
    hipMalloc( &d_graph, size );
    initializeGraph<<<1, V * V>>>( d_graph, V );
    hipDeviceSynchronize();

    int* h_graph = new int[V * V];
    hipMemcpy( h_graph, d_graph, size, hipMemcpyDeviceToHost );

    // Display the graph
    for( int i = 0 ; i < V ; i++ ){
        for( int j = 0 ; j < V ; j++ ){
            cout << h_graph[i * V + j] << " ";
        }
        cout << endl;
    }

    hipFree( d_graph );
    delete[] h_graph;
    return 0;
}
