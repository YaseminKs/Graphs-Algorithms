// Connected Graph in CUDA

#include <stdio.h>
#include <hip/hip_runtime.h>

__global__ void checkConnectivity( int *graph, int *visited, int V ){
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if( idx < V && visited[idx] ){
        for( int j = 0 ; j < V ; j++ ){
            if( graph[idx * V + j] && !visited[j] )
                visited[j] = 1;
        }
    }
}

int main(){
    int V = 4;
    int graph[16] = { 0, 1, 0, 0,
                     1, 0, 1, 0,
                     0, 1, 0, 1,
                     0, 0, 1, 0 };
    int visited[4] = { 1, 0, 0, 0 };

    int *d_graph, *d_visited;
    hipMalloc( &d_graph, V * V * sizeof( int ) );
    hipMalloc( &d_visited, V * sizeof( int ) );

    hipMemcpy( d_graph, graph, V * V * sizeof( int ), hipMemcpyHostToDevice );
    hipMemcpy( d_visited, visited, V * sizeof( int ), hipMemcpyHostToDevice );

    checkConnectivity<<<1, V>>>(d_graph, d_visited, V);

    hipMemcpy( visited, d_visited, V * sizeof( int ), hipMemcpyDeviceToHost );

    bool connected = true;
    for( int i = 0 ; i < V ; i++ )
        if( !visited[i]) connected = false;

    printf( "Is graph connected? %s\n", connected ? "Yes" : "No" );

    hipFree( d_graph );
    hipFree( d_visited );
    
  return 0;
}
