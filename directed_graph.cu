// Parallel Directed Graph Initialization
// GPU Parallel

#include <hip/hip_runtime.h>
#include <iostream>

using namespace std;

__global__ void initGraph( int* graph, int V ){
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if( idx < V * V ){
        graph[idx] = 0;
    }
}

__global__ void addEdge( int* graph, int V, int u, int v ){
    graph[u * V + v] = 1;
}

int main(){
    int V = 3;
    int* d_graph;
    size_t size = V * V * sizeof( int );
    hipMalloc( &d_graph, size );
    initGraph<<<1, V * V>>>( d_graph, V );
    hipDeviceSynchronize();

    // Add directed edges (0->1, 1->2)
    addEdge<<<1, 1>>>( d_graph, V, 0, 1 );
    addEdge<<<1, 1>>>( d_graph, V, 1, 2 );
    hipDeviceSynchronize();

    int* h_graph = new int[V * V];
    hipMemcpy( h_graph, d_graph, size, hipMemcpyDeviceToHost );

    // Display graph
    for( int i = 0 ; i < V ; i++ ){
        for( int j = 0 ; j < V ; j++ ){
            cout << h_graph[i * V + j] << " ";
        }
        cout << endl;
    }

    hipFree( d_graph );
    delete[] h_graph;
    return 0;
}
