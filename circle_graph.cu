
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void cycleGraph( int n ){
    int i = threadIdx.x;
    printf( "%d -> %d\n", i, ( i+1 )%n );
}

int main(){
    int n = 5;
    cycleGraph<<<1, n>>>( n );
    hipDeviceSynchronize();
    return 0;
}
