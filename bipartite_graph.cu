// Simplified Version

#include <iostream>
#include <hip/hip_runtime.h>

__global__ void findMatching( int *adjMatrix, int *matchesA, int *matchesB, int numNodesA, int numNodesB ){
    int idx = threadIdx.x + blockIdx.x * blockDim.x;

    // Ensure we're within bounds of A and B
    if( idx < numNodesA ){
        for( int j = 0 ; j < numNodesB ; ++j ){
            // Check if an edge exists between A[idx] and B[j]
            if( adjMatrix[idx * numNodesB + j] == 1 ){
                // Check if B[j] is not already matched, then match A[idx] with B[j]
                if( matchesB[j] == -1 ){
                    matchesA[idx] = j;
                    matchesB[j] = idx;
                    return;
                }
            }
        }
    }
}

int main(){
    int numNodesA = 4;  // Number of nodes in set A
    int numNodesB = 4;  // Number of nodes in set B

    // Example adjacency matrix (1 means an edge exists, 0 means no edge)
    int adjMatrix[] = {
        1, 0, 1, 0,
        0, 1, 1, 0,
        1, 1, 0, 1,
        0, 0, 1, 1
    };

    int *d_adjMatrix, *d_matchesA, *d_matchesB;
    int *matchesA = new int[numNodesA];
    int *matchesB = new int[numNodesB];

    // Initialize match arrays (-1 means no match)
    for( int i = 0 ; i < numNodesA ; ++i ){
        matchesA[i] = -1;
    }
  
    for( int i = 0 ; i < numNodesB ; ++i ){
        matchesB[i] = -1;
    }

    // Allocate memory on the device
    hipMalloc( &d_adjMatrix, numNodesA * numNodesB * sizeof( int ) );
    hipMalloc( &d_matchesA, numNodesA * sizeof( int ) );
    hipMalloc( &d_matchesB, numNodesB * sizeof( int ) );

    // Copy data to device
    hipMemcpy( d_adjMatrix, adjMatrix, numNodesA * numNodesB * sizeof( int ), hipMemcpyHostToDevice );
    hipMemcpy( d_matchesA, matchesA, numNodesA * sizeof( int ), hipMemcpyHostToDevice );
    hipMemcpy( d_matchesB, matchesB, numNodesB * sizeof( int ), hipMemcpyHostToDevice );

    // Define block and grid sizes
    int blockSize = 256;  // Number of threads per block
    int gridSize = ( numNodesA + blockSize - 1 ) / blockSize;  // Number of blocks needed

    // Launch the kernel to find matching
    findMatching<<<gridSize, blockSize>>>( d_adjMatrix, d_matchesA, d_matchesB, numNodesA, numNodesB );

    // Copy results back to host
    hipMemcpy( matchesA, d_matchesA, numNodesA * sizeof( int ), hipMemcpyDeviceToHost );
    hipMemcpy( matchesB, d_matchesB, numNodesB * sizeof( int ), hipMemcpyDeviceToHost );

    // Display the result (matches)
    std::cout << "Matches from A to B:\n";
    for( int i = 0 ; i < numNodesA ; ++i ){
        if( matchesA[i] != -1 ){
            std::cout << "A[" << i << "] is matched to B[" << matchesA[i] << "]\n";
        }else{
            std::cout << "A[" << i << "] has no match\n";
        }
    }

    // Cleanup
    hipFree( d_adjMatrix );
    hipFree( d_matchesA );
    hipFree( d_matchesB );
    delete[] matchesA;
    delete[] matchesB;

    return 0;
}
